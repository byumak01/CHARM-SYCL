#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstdint>
#include <cstdlib>
#include <utility>
#include "common.hpp"

#define _(expr)                                                                     \
    ({                                                                              \
        auto const err__ = (expr);                                                  \
        if (err__ != hipSuccess) {                                                 \
            show_msg_fmt("CUDA Error: %s: %s\n", #expr, hipGetErrorString(err__)); \
            app_exit(1);                                                            \
        }                                                                           \
    })

__global__ void vecadd_kernel(float const* __restrict a, float const* __restrict b,
                              float* __restrict c, size_t n) {
    auto const i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void run_benchmark(float const* __restrict a, float const* __restrict b, float* __restrict c,
                   size_t n, size_t n_loop, size_t, uint64_t* runtime) {
    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;
    hipStream_t stream;

    _(hipMalloc(&d_a, n * sizeof(float)));
    _(hipMalloc(&d_b, n * sizeof(float)));
    _(hipMalloc(&d_c, n * sizeof(float)));
    _(hipStreamCreate(&stream));

    _(hipMemcpyAsync(d_a, a, n * sizeof(float), hipMemcpyHostToDevice, stream));
    _(hipMemcpyAsync(d_b, b, n * sizeof(float), hipMemcpyHostToDevice, stream));
    _(hipStreamSynchronize(stream));

    auto const nb = (n + 255) / 256;
    auto const nt = std::min<size_t>(n, 256);

    for (int loop = 0; loop < n_loop; loop++) {
        auto const t_start = std::chrono::high_resolution_clock::now();

        vecadd_kernel<<<nb, nt, 0, stream>>>(d_a, d_b, d_c, n);
        _(hipStreamSynchronize(stream));

        auto const t_end = std::chrono::high_resolution_clock::now();
        runtime[loop] =
            std::chrono::duration_cast<std::chrono::nanoseconds>(t_end - t_start).count();
    }

    _(hipMemcpyAsync(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost, stream));
    _(hipStreamSynchronize(stream));

    _(hipFree(d_a));
    _(hipFree(d_b));
    _(hipFree(d_c));
}
