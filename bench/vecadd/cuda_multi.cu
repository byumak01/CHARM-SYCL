#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstdint>
#include <cstdlib>
#include <functional>
#include <numeric>
#include <utility>
#include <vector>
#include "common.hpp"

#define _(expr)                                                                     \
    ({                                                                              \
        auto const err__ = (expr);                                                  \
        if (err__ != hipSuccess) {                                                 \
            show_msg_fmt("CUDA Error: %s: %s\n", #expr, hipGetErrorString(err__)); \
            app_exit(1);                                                            \
        }                                                                           \
    })

__global__ void vecadd_kernel(float const* __restrict a, float const* __restrict b,
                              float* __restrict c, size_t n) {
    auto const i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void run_benchmark(float const* __restrict a, float const* __restrict b, float* __restrict c,
                   size_t n, size_t n_loop, size_t n_chunk, uint64_t* runtime) {
    int n_dev;
    _(hipGetDeviceCount(&n_dev));

    std::vector<size_t> offsets, sizes;
    split_vecadd(n, offsets, sizes, n_chunk);

    std::vector<float*> d_a(n_chunk);
    std::vector<float*> d_b(n_chunk);
    std::vector<float*> d_c(n_chunk);
    std::vector<hipStream_t> stream(n_chunk);

    for (size_t i = 0; i < n_chunk; i++) {
        auto const d_off = offsets.at(i);
        auto const d_size = sizes.at(i);
        auto const dev = i % n_dev;

        _(hipSetDevice(dev));
        _(hipMalloc(&d_a.at(i), d_size * sizeof(float)));
        _(hipMalloc(&d_b.at(i), d_size * sizeof(float)));
        _(hipMalloc(&d_c.at(i), d_size * sizeof(float)));

        _(hipStreamCreate(&stream.at(i)));
        _(hipMemcpyAsync(d_a.at(i), a + d_off, d_size * sizeof(float), hipMemcpyHostToDevice,
                          stream.at(i)));
        _(hipMemcpyAsync(d_b.at(i), b + d_off, d_size * sizeof(float), hipMemcpyHostToDevice,
                          stream.at(i)));
    }
    for (int i = 0; i < n_chunk; i++) {
        _(hipStreamSynchronize(stream.at(i)));
    }

    for (int loop = 0; loop < n_loop; loop++) {
        auto const t_start = std::chrono::high_resolution_clock::now();

        for (size_t i = 0; i < n_chunk; i++) {
            auto const d_size = sizes.at(i);
            auto const dev = i % n_dev;
            auto const nb = (d_size + 255) / 256;
            auto const nt = std::min<size_t>(d_size, 256);

            _(hipSetDevice(dev));
            vecadd_kernel<<<nb, nt, 0, stream.at(i)>>>(d_a.at(i), d_b.at(i), d_c.at(i), d_size);
        }
        for (int i = 0; i < n_chunk; i++) {
            _(hipStreamSynchronize(stream.at(i)));
        }

        auto const t_end = std::chrono::high_resolution_clock::now();
        runtime[loop] =
            std::chrono::duration_cast<std::chrono::nanoseconds>(t_end - t_start).count();
    }

    for (size_t i = 0; i < n_chunk; i++) {
        auto const d_off = offsets.at(i);
        auto const d_size = sizes.at(i);

        _(hipMemcpyAsync(c + d_off, d_c.at(i), d_size * sizeof(float), hipMemcpyDeviceToHost,
                          stream.at(i)));
    }
    for (size_t i = 0; i < n_chunk; i++) {
        _(hipStreamSynchronize(stream.at(i)));
        _(hipStreamDestroy(stream.at(i)));
        _(hipFree(d_a.at(i)));
        _(hipFree(d_b.at(i)));
        _(hipFree(d_c.at(i)));
    }
}
